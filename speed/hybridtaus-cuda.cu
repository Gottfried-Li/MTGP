#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <util.h>

// S1, S2, S3, and M are all constants, and z is part of the
// private per-thread generator state.
__device__ unsigned TausStep(unsigned &z, int S1, int S2, int S3, unsigned M)
{
    unsigned b=(((z << S1) ^ z) >> S2);
    return z = (((z & M) << S3) ^ b);
}

// A and C are constants
__device__ unsigned LCGStep(unsigned &z, unsigned A, unsigned C)
{
    return z=(A*z+C);
}

__device__ float HybridTaus(unsigned& z1,
			    unsigned& z2,
			    unsigned& z3,
			    unsigned& z4)
{
    // Combined period is lcm(p1,p2,p3,p4)~ 2^121
    uint32_t r = (
	TausStep(z1, 13, 19, 12, 4294967294UL) ^  // p1=2^31-1
	TausStep(z2, 2, 25, 4, 4294967288UL) ^    // p2=2^30-1
	TausStep(z3, 3, 11, 17, 4294967280UL) ^   // p3=2^28-1
	LCGStep(z4, 1664525, 1013904223UL)        // p4=2^32
	);
#if defined(FLOAT_MASK)
    r = (r >> 9) | 0x3f800000U;
    return __int_as_float(r) - 1.0f;
#else
    return 2.3283064365387e-10 * r;
#endif
}

__global__ void hybrid_kernel(float* d_data, int size)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    unsigned z1 = bid * 13 + tid + 1;
    unsigned z2 = tid * 29 + bid + 9;
    unsigned z3 = bid * 7 + tid * 97 + 8;
    unsigned z4 = bid * 19937 + tid * 607 + 2;

    for (int i = 0; i < size; i += blockDim.x) {
	d_data[size * bid + i + tid] = HybridTaus(z1, z2, z3, z4);
    }
}

__global__ void hybrid_reduce(float* d_data, int size)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    unsigned z1 = bid * 13 + tid + 1;
    unsigned z2 = tid * 29 + bid + 9;
    unsigned z3 = bid * 7 + tid * 97 + 8;
    unsigned z4 = bid * 19937 + tid * 607 + 2;
    float sum = 0;

    for (int i = 0; i < size; i += blockDim.x) {
	sum += HybridTaus(z1, z2, z3, z4);
    }
    d_data[blockDim.x * bid + tid] = sum;
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] num_data number of data to be generated.
 */
void make_hybrid_random(int num_data,
			int block_num) {
    float* d_data;
    float* h_data;
    hipError_t e;

    printf("generating float random numbers.\n");
    ccudaMalloc((void**)&d_data, sizeof(float) * num_data);
    /* cutCreateTimer(&timer); */
    float elapsed_time_ms=0.0f;
    hipEvent_t start, stop;
    ccudaEventCreate(&start);
    ccudaEventCreate(&stop);
    h_data = (float *) malloc(sizeof(float) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    /* ccutStartTimer(timer); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    hybrid_kernel<<< block_num, THREAD_NUM>>>(d_data, num_data / block_num);
    ccudaEventRecord(stop, 0);
    ccudaEventSynchronize(stop);
    ccudaThreadSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaMemcpy(h_data,
		d_data,
		sizeof(float) * num_data,
		hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer);*/
    ccudaEventElapsedTime(&elapsed_time_ms, start, stop);

    print_float_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", elapsed_time_ms);
    printf("Samples per second: %E \n", num_data / (elapsed_time_ms * 0.001));
    /* CUT_SAFE_CALL(cutDeleteTimer(timer));*/
    ccudaEventDestroy(start);
    ccudaEventDestroy(stop);
    //free memories
    free(h_data);
    ccudaFree(d_data);
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] num_data number of data to be generated.
 */
void make_hybrid_reduce(int num_data,
			int block_num) {
    float* d_data;
    float* h_data;
    hipError_t e;

    printf("generating float random numbers.\n");
    ccudaMalloc((void**)&d_data, sizeof(float) * block_num * THREAD_NUM);
    /* CUT_SAFE_CALL(cutCreateTimer(&timer)); */
    float elapsed_time_ms=0.0f;
    hipEvent_t start, stop;
    ccudaEventCreate(&start);
    ccudaEventCreate(&stop);

    h_data = (float *) malloc(sizeof(float) * block_num * THREAD_NUM);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    /* CUT_SAFE_CALL(cutStartTimer(timer)); */
    ccudaEventRecord(start, 0);

    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    hybrid_reduce<<< block_num, THREAD_NUM>>>(d_data, num_data / block_num);
    ccudaEventRecord(stop, 0);
    ccudaEventSynchronize(stop);
    ccudaThreadSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventElapsedTime(&elapsed_time_ms, start, stop);
    ccudaMemcpy(h_data, d_data, sizeof(float) * block_num * THREAD_NUM,
		hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", elapsed_time_ms);
    printf("Samples per second: %E \n", num_data / (elapsed_time_ms * 0.001));
    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(stop);
    //free memories
    free(h_data);
    ccudaFree(d_data);
}

int main(int argc, char** argv)
{
    int num_data = 1;
    int block_num;
    int num_unit;
    int r;

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (block_num < 1) {
	    printf("%s block_num should be larger than 1\n",
		   argv[0], BLOCK_NUM_MAX);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	printf("%s number_of_block number_of_output\n", argv[0]);
	return 1;
    }
    num_unit = THREAD_NUM * block_num;
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_hybrid_random(num_data, block_num);
    make_hybrid_reduce(num_data, block_num);
}
