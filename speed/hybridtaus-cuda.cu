#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <util.h>

#define BLOCK_NUM_MAX 1000
#define THREAD_NUM 256
// S1, S2, S3, and M are all constants, and z is part of the
// private per-thread generator state.
__device__ unsigned TausStep(unsigned &z, int S1, int S2, int S3, unsigned M)
{
    unsigned b=(((z << S1) ^ z) >> S2);
    return z = (((z & M) << S3) ^ b);
}

// A and C are constants
__device__ unsigned LCGStep(unsigned &z, unsigned A, unsigned C)
{
    return z=(A*z+C);
}

__device__ float HybridTaus(unsigned& z1,
			    unsigned& z2,
			    unsigned& z3,
			    unsigned& z4)
{
    // Combined period is lcm(p1,p2,p3,p4)~ 2^121
    return 2.3283064365387e-10 * (              // Periods
	TausStep(z1, 13, 19, 12, 4294967294UL) ^  // p1=2^31-1
	TausStep(z2, 2, 25, 4, 4294967288UL) ^    // p2=2^30-1
	TausStep(z3, 3, 11, 17, 4294967280UL) ^   // p3=2^28-1
	LCGStep(z4, 1664525, 1013904223UL)        // p4=2^32
	);
}

__global__ void hybrid_kernel(float* d_data, int size)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    unsigned z1 = bid * 13 + tid + 1;
    unsigned z2 = tid * 29 + bid + 9;
    unsigned z3 = bid * 7 + tid * 97 + 8;
    unsigned z4 = bid * 19937 + tid * 607 + 2;

    for (int i = 0; i < size; i += blockDim.x) {
	d_data[size * bid + i + tid] = HybridTaus(z1, z2, z3, z4);
    }
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] num_data number of data to be generated.
 */
void make_hybrid_random(int num_data,
			int block_num) {
    float* d_data;
    unsigned int timer = 0;
    float* h_data;
    hipError_t e;
    float gputime;

    printf("generating float random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(float) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (float *) malloc(sizeof(float) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    hybrid_kernel<<< block_num, THREAD_NUM>>>(d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(float) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_float_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

int main(int argc, char** argv)
{
    int num_data = 1;
    int block_num;
    int num_unit;
    int r;

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (block_num < 1 || block_num > BLOCK_NUM_MAX) {
	    printf("%s block_num should be between 1 and %d\n",
		   argv[0], BLOCK_NUM_MAX);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	CUT_DEVICE_INIT(argc, argv);
	printf("%s number_of_block number_of_output\n", argv[0]);
	return 1;
    }
    CUT_DEVICE_INIT(argc, argv);
    num_unit = THREAD_NUM * block_num;
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_hybrid_random(num_data, block_num);

    //finalize
#ifdef NEED_PROMPT
    CUT_EXIT(argc, argv);
#endif
}
